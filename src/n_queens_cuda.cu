#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>

#include "macro.h"
#include "n_queens.h"
#include "utils.h"

inline int get_block_size(long long size, int block_size) { return (size + block_size - 1) / block_size; }

__global__ void n_queens(int N, int *tot, long long *partial_sum, long long cnt) {
    const long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt) {
        int last = (1 << N) - 1;
        long long sum = 0;
        __shared__ int stack[48 * 256];
        const int bottom = threadIdx.x / 32 * 32 * STACKSIZE + threadIdx.x % 32;
        int top = bottom;

        int cur = tot[tid * 3];
        int left = tot[tid * 3 + 1];
        int right = tot[tid * 3 + 2];
        int valid_pos = last & (~(cur | left | right));

        if (valid_pos == 0) return;

        stack[top] = cur;
        stack[top + 32] = left;
        stack[top + 64] = right;
        stack[top + 96] = valid_pos;
        top += 128;

        while (top != bottom) {
            valid_pos = stack[top - 32];
            right = stack[top - 64];
            left = stack[top - 96];
            cur = stack[top - 128];

            int p = valid_pos & (-valid_pos);
            valid_pos -= p;
            stack[top - 32] = valid_pos;
            top -= (valid_pos == 0 ? 128 : 0);

            cur = cur | p;
            left = (left | p) << 1;
            right = (right | p) >> 1;
            valid_pos = last & (~(cur | left | right));

            if (valid_pos == 0) {
                continue;
            }

            p = cur ^ last;
            if((p & (p - 1)) == 0) {
                sum++; // only has one valid position
                continue;
            }

            stack[top] = cur;
            stack[top + 32] = left;
            stack[top + 64] = right;
            stack[top + 96] = valid_pos;
            top += 128;
        }

        partial_sum[tid] = sum;
    }
}

long long cuda_n_queens(int N, int rows) {
    struct timeval start, end;
    long long sum = 0;
    vector<int> tot;

    // 1. get total subproblems.
    gettimeofday(&start, NULL);
    partial_n_queens(N, 0, 0, 0, tot, rows);

    if (N & 0x1) {
        partial_n_queens_for_odd(N, 0, 0, 0, tot, rows);
    }

    long long cnt = tot.size() / 3;
    vector<long long> partial_sum(cnt);
    gettimeofday(&end, NULL);

    print_with_time("Use %.2fms to generate %lld subproblems!\n", time_diff_ms(start, end), cnt);

    int gpu_num = 0;
    hipGetDeviceCount(&gpu_num);
    if (gpu_num == 0) {
        printf("Failed to find any gpu!\n");
        return -1;
    }

    // 2. divide total subproblems to different trunks
    vector<long long> new_cnt(gpu_num), start_pos(gpu_num);
    long long total = 0;
    if (gpu_num == 8) {
        float ratio[8] = {0.18, 0.16, 0.13, 0.12, 0.11, 0.1, 0.1, 0.1};
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = cnt * ratio[i];
            start_pos[i] = total;
            total += new_cnt[i];
        }
    } else if (gpu_num == 4) {
        float ratio[4] = {0.34, 0.25, 0.21, 0.2};
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = cnt * ratio[i];
            start_pos[i] = total;
            total += new_cnt[i];
        }
    } else if (gpu_num == 2) {
        float ratio[2] = {0.59, 0.41};
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = cnt * ratio[i];
            start_pos[i] = total;
            total += new_cnt[i];
        }
    } else {
        long long partial_cnt = cnt / gpu_num;
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = partial_cnt;
            start_pos[i] = total;
            total += partial_cnt;
        }
    }
    new_cnt[gpu_num - 1] = cnt - total;
    start_pos[gpu_num - 1] = total;

    // 3. use different gpu to process each trunk
#pragma omp parallel num_threads(gpu_num)
    {
        int idx = omp_get_thread_num();
        CU_SAFE_CALL(hipSetDevice(idx));

        long long cnt = new_cnt[idx];

        print_with_time("gpu [%d] start job, with %lld subproblems.\n", idx, cnt);

        int *cuda_tot;
        CU_SAFE_CALL(hipMalloc(&cuda_tot, sizeof(int) * cnt * 3));
        CU_SAFE_CALL(hipMemcpy(cuda_tot, tot.data() + start_pos[idx] * 3, sizeof(int) * cnt * 3, hipMemcpyHostToDevice));

        long long *cuda_partial_sum;
        CU_SAFE_CALL(hipMalloc(&cuda_partial_sum, sizeof(long long) * cnt));
        CU_SAFE_CALL(hipMemset(cuda_partial_sum, 0, sizeof(long long) * cnt));

        dim3 dimBlock(CU1DBLOCK);
        dim3 dimGrid(get_block_size(cnt, CU1DBLOCK));

        n_queens<<<dimGrid, dimBlock>>>(N, cuda_tot, cuda_partial_sum, cnt);

        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("kernel error: %s\n", hipGetErrorString(err));
        }

        CU_SAFE_CALL(hipMemcpy(partial_sum.data() + start_pos[idx], cuda_partial_sum, sizeof(long long) * cnt, hipMemcpyDeviceToHost));

        CU_SAFE_CALL(hipFree(cuda_tot));
        CU_SAFE_CALL(hipFree(cuda_partial_sum));
        print_with_time("gpu [%d] finish job.\n", idx);
    }

    for (long long i = 0; i < cnt; i++) {
        sum += partial_sum[i] * 2;
    }

    return sum;
}
