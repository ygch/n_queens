#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "macro.h"
#include "n_queens.h"

inline int get_block_size(int size, int block_size) { return (size + block_size - 1) / block_size; }

__device__ void n_queens_device(int N, int cur, int left, int right, long long *sum) {
    int last = (1 << N) - 1;
    if (cur == last) {
        (*sum)++;
        return;
    }

    int valid_pos = last & (~(cur | left | right));
    while (valid_pos) {
        int p = valid_pos & (-valid_pos);
        valid_pos -= p;
        n_queens_device(N, cur | p, (left | p) << 1, (right | p) >> 1, sum);
    }
}

__device__ long long n_queens_device_iterative(int N, int cur, int left, int right) {
    long long sum = 0;
    int last = (1 << N) - 1;
    int stack[192];
    int top = 0;
    stack[top++] = cur;
    stack[top++] = left;
    stack[top++] = right;

    while (top != 0) {
        right = stack[--top];
        left = stack[--top];
        cur = stack[--top];

        if (cur == last) {
            sum++;
            continue;
        }

        int valid_pos = last & (~(cur | left | right));
        while (valid_pos) {
            int p = valid_pos & (-valid_pos);
            valid_pos -= p;
            stack[top++] = cur | p;
            stack[top++] = (left | p) << 1;
            stack[top++] = (right | p) >> 1;
        }
    }

    return sum;
}

__global__ void n_queens(int N, int *tot, long long *partial_sum, int cnt) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt) {
        partial_sum[tid] = n_queens_device_iterative(N, tot[tid * 3], tot[tid * 3 + 1], tot[tid * 3 + 2]);
    }
}

long long cuda_n_queens(int N, int level) {
    long long sum = 0;
    vector<int> tot;

    partial_n_queens(N, 0, 0, 0, tot, 0, level);

    int cnt = tot.size() / 3;

    if (N & 0x1) {
        partial_n_queens_for_odd(N, 0, 0, 0, tot, 0, level);
    }

    int new_cnt = tot.size() / 3;
    vector<long long> partial_sum(new_cnt);

    // random_shuffle(tot.data(), cnt);
    // random_shuffle(tot.data() + cnt * 3, new_cnt - cnt);

    int *cuda_tot;
    long long *cuda_partial_sum;
    CU_SAFE_CALL(hipMalloc(&cuda_tot, sizeof(int) * new_cnt * 3));
    CU_SAFE_CALL(hipMalloc(&cuda_partial_sum, sizeof(long long) * new_cnt));

    CU_SAFE_CALL(hipMemcpy(cuda_tot, tot.data(), sizeof(int) * new_cnt * 3, hipMemcpyHostToDevice));
    CU_SAFE_CALL(hipMemcpy(cuda_partial_sum, partial_sum.data(), sizeof(long long) * new_cnt, hipMemcpyHostToDevice));

    dim3 dimBlock(CU1DBLOCK);
    dim3 dimGrid(get_block_size(new_cnt, CU1DBLOCK));

    printf("total size %d, block size %d, grid size %d\n", new_cnt, CU1DBLOCK, get_block_size(new_cnt, CU1DBLOCK));

    n_queens<<<dimGrid, dimBlock>>>(N, cuda_tot, cuda_partial_sum, new_cnt);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("kernel error: %s\n", hipGetErrorString(err));
    }

    CU_SAFE_CALL(hipMemcpy(partial_sum.data(), cuda_partial_sum, sizeof(long long) * new_cnt, hipMemcpyDeviceToHost));

    for (int i = 0; i < cnt; i++) {
        sum += partial_sum[i] * 2;
    }

    for (int i = cnt; i < new_cnt; i++) {
        sum += partial_sum[i];
    }

    CU_SAFE_CALL(hipFree(cuda_tot));
    CU_SAFE_CALL(hipFree(cuda_partial_sum));

    return sum;
}
