#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <omp.h>

#include "macro.h"
#include "n_queens.h"
#include "utils.h"

inline int get_block_size(long long size, int block_size) { return (size + block_size - 1) / block_size; }

__global__ void n_queens(int N, int *tot, long long *partial_sum, long long cnt) {
    const long long tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int last = (1 << N) - 1;

    if (tid < cnt) {
        long long sum = 0;
        const int bottom = (threadIdx.x / 32) * 32 * STACKSIZE + threadIdx.x % 32;
        int top = bottom;

        int cur = tot[tid * 3];
        int left = tot[tid * 3 + 1];
        int right = tot[tid * 3 + 2];
        int valid_pos = last & ~cur & ~left & ~right;

        asm(".reg .s32 base_addr, top, tmp, tmp2;\n\t"
            ".reg .s64 ltmp, ltmp2;\n\t"
            ".reg .pred p, q, z;\n\t"
            ".shared .align 16 .b8 stack[49152];\n\t"

            " mov.u32 top, %5;\n\t"
            " mov.u32 base_addr, stack;\n\t"
            " mad.lo.s32 tmp2, top, 16, base_addr;\n\t"
            " st.shared.v4.u32 [tmp2], {%1, %2, %3, %4};\n\t"               // stack[top] = {cur, left, right, valid_pos}
            " setp.eq.s32 p, %4, 0;\n\t"                                    // valid_pos == 0
            " @p bra FINISH;\n\t"                                           // return;
            " add.s32 top, top, 32;\n\t"                                    // top += 32

            " LOOP:\n\t"
            " setp.eq.s32 p, top, %6;\n\t"                                  // top == bottom
            " @p bra FINISH;\n\t"                                           // done

            " mad.lo.s32 tmp2, top, 16, base_addr;\n\t"
            " ld.shared.v4.u32 {%1, %2, %3, %4}, [tmp2 + -512];\n\t"        // {cur, left, right, valid_pos} = stack[top - 32]

            " neg.s32 tmp, %4;\n\t"                                         // p = -valid_pos
            " and.b32 tmp, %4, tmp;\n\t"                                    // p = valid_pos & (-valid_pos)
            " sub.s32 %4, %4, tmp;\n\t"                                     // valid_pos -= p
            " st.shared.s32 [tmp2 + -500], %4;\n\t"                         // stack[top - 32] = valid_pos
            " setp.eq.s32 p, %4, 0;\n\t"                                    // p = (valid_pos == 0)
            " selp.b32 tmp2, 32, 0, p;\n\t"                                 // tmp2 = (p==1 ? 32 : 0)
            " sub.s32 top, top, tmp2;\n\t"                                  // top -= 32

            " or.b32 %1, %1, tmp;\n\t"                                      // cur = cur | p
            " or.b32 %2, %2, tmp;\n\t"                                      // left = left | p
            " shl.b32 %2, %2, 1;\n\t"                                       // left = left << 1
            " or.b32 %3, %3, tmp;\n\t"                                      // right = right | p
            " shr.b32 %3, %3, 1;\n\t"                                       // right = right >> 1
            " lop3.b32 tmp, %1, %2, %3, 0x1;\n\t"                           // tmp = ~cur & ~left & ~right;
            " and.b32 %4, %7, tmp;\n\t"                                     // valid_pos = last & tmp
            " popc.b32 tmp, %1;\n\t"                                        // tmp = popc(cur)
            " setp.eq.s32 p, tmp, %8;\n\t"                                  // popc(cur) == N - 1
            " setp.eq.s32 q, %4, 0;\n\t"                                    // valid_pos == 0
            " or.pred z, p, q;\n\t"                                         // valid_pos == 0 || popc(cur) == N - 1

            " popc.b32 tmp, %4;\n\t"                                        // tmp = popc(valid_pos)
            " cvt.s64.s32 ltmp, tmp;\n\t"
            " cvt.s64.s32 ltmp2, 0;\n\t"
            " selp.s64 ltmp, ltmp, ltmp2, z;\n\t"                           // ltmp = (z == 1 ? ltmp : ltmp2)
            " add.s64 %0, %0, ltmp;\n\t"                                    // sum += 1

            " @!z mad.lo.s32 tmp2, top, 16, base_addr;\n\t"
            " @!z st.shared.v4.u32 [tmp2], {%1, %2, %3, %4};\n\t"           // stack[top] = {cur, left, right, valid_pos}
            " @!z add.s32 top, top, 32;\n\t"                                // top += 32
            " bra.uni LOOP;\n\t"

            " FINISH:\n\t"
            :"+l"(sum), "+r"(cur), "+r"(left), "+r"(right), "+r"(valid_pos) // output
            :"r"(top), "r"(bottom), "r"(last), "r"(N - 1)                   // input
        );

        partial_sum[tid] = sum;
    }
}

long long cuda_n_queens(int N, int rows) {
    struct timeval start, end;
    long long sum = 0;
    vector<int> tot;

    // 1. get total subproblems.
    gettimeofday(&start, NULL);
    partial_n_queens(N, 0, 0, 0, tot, rows);

    if (N & 0x1) {
        partial_n_queens_for_odd(N, 0, 0, 0, tot, rows);
    }

    long long cnt = tot.size() / 3;
    vector<long long> partial_sum(cnt);
    gettimeofday(&end, NULL);

    print_with_time("Use %.2fms to generate %lld subproblems!\n", time_diff_ms(start, end), cnt);

    int gpu_num = 0;
    hipGetDeviceCount(&gpu_num);
    if (gpu_num == 0) {
        printf("Failed to find any gpu!\n");
        return -1;
    }

    // 2. divide total subproblems to different trunks
    vector<long long> new_cnt(gpu_num), start_pos(gpu_num);
    long long total = 0;
    if (gpu_num == 8) {
        //float ratio[8] = {0.198, 0.156, 0.128, 0.117, 0.105, 0.100, 0.098, 0.098};
        float ratio[8] = {0.18, 0.16, 0.13, 0.12, 0.11, 0.100, 0.10, 0.10};
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = cnt * ratio[i];
            start_pos[i] = total;
            total += new_cnt[i];
        }
    } else if (gpu_num == 4) {
        float ratio[4] = {0.34, 0.25, 0.21, 0.2};
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = cnt * ratio[i];
            start_pos[i] = total;
            total += new_cnt[i];
        }
    } else if (gpu_num == 2) {
        float ratio[2] = {0.59, 0.41};
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = cnt * ratio[i];
            start_pos[i] = total;
            total += new_cnt[i];
        }
    } else {
        long long partial_cnt = cnt / gpu_num;
        for (int i = 0; i < gpu_num - 1; i++) {
            new_cnt[i] = partial_cnt;
            start_pos[i] = total;
            total += partial_cnt;
        }
    }
    new_cnt[gpu_num - 1] = cnt - total;
    start_pos[gpu_num - 1] = total;

    // 3. use different gpu to process each trunk
#pragma omp parallel num_threads(gpu_num)
    {
        int idx = omp_get_thread_num();
        CU_SAFE_CALL(hipSetDevice(idx));

        long long cnt = new_cnt[idx];

        print_with_time("gpu [%d] start job, with %lld subproblems.\n", idx, cnt);

        int *cuda_tot;
        CU_SAFE_CALL(hipMalloc(&cuda_tot, sizeof(int) * cnt * 3));
        CU_SAFE_CALL(hipMemcpy(cuda_tot, tot.data() + start_pos[idx] * 3, sizeof(int) * cnt * 3, hipMemcpyHostToDevice));

        long long *cuda_partial_sum;
        CU_SAFE_CALL(hipMalloc(&cuda_partial_sum, sizeof(long long) * cnt));
        CU_SAFE_CALL(hipMemset(cuda_partial_sum, 0, sizeof(long long) * cnt));

        dim3 dimBlock(CU1DBLOCK);
        dim3 dimGrid(get_block_size(cnt, CU1DBLOCK));

        n_queens<<<dimGrid, dimBlock>>>(N, cuda_tot, cuda_partial_sum, cnt);

        hipError_t err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            printf("kernel error: %s\n", hipGetErrorString(err));
        }

        CU_SAFE_CALL(hipMemcpy(partial_sum.data() + start_pos[idx], cuda_partial_sum, sizeof(long long) * cnt, hipMemcpyDeviceToHost));

        CU_SAFE_CALL(hipFree(cuda_tot));
        CU_SAFE_CALL(hipFree(cuda_partial_sum));
        print_with_time("gpu [%d] finish job.\n", idx);
    }

    for (long long i = 0; i < cnt; i++) {
        sum += partial_sum[i] * 2;
    }

    return sum;
}
