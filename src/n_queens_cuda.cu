#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "macro.h"
#include "n_queens.h"

inline int get_block_size(long long size, int block_size) { return (size + block_size - 1) / block_size; }

__global__ void n_queens(int N, int *tot, long long *partial_sum, long long cnt) {
    long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < cnt) {
        int last = (1 << N) - 1;
        int mask = (1 << 21) - 1;
        long long sum = 0;
        __shared__ long long stack_clr[21 * 192];
        __shared__ int stack_valid_pos[22 * 192];
        int idx = threadIdx.x / 32 * 32 * 21 + threadIdx.x % 32;

        int top = idx;

        long long cur = tot[tid * 3];
        long long left = tot[tid * 3 + 1];
        long long right = tot[tid * 3 + 2];
        int valid_pos = last & (~(cur | left | right));

        if (valid_pos == 0) return;

        stack_clr[top] = cur | (left << 21) | (right << 42);
        stack_valid_pos[top] = valid_pos;
        top += 32;

        while (top != idx) {
            valid_pos = stack_valid_pos[top - 32];

            long long val = stack_clr[top - 32];
            cur = val & mask;
            val >>= 21;
            left = val & mask;
            val >>= 21;
            right = val;

            if(tid == 0) {
                printf("top %d, [%d, %lld, %lld, %lld]\n", top, valid_pos, cur, left, right);
            }

            int p = valid_pos & (-valid_pos);
            valid_pos -= p;

            if (valid_pos == 0) {
                top -= 32;
            } else {
                stack_valid_pos[top - 32] = valid_pos;
            }

            cur = cur | p;
            if (cur == last) {
                sum++;
                continue;
            }

            left = (left | p) << 1;
            right = (right | p) >> 1;
            valid_pos = last & (~(cur | left | right));

            if (valid_pos == 0) {
                continue;
            }

            stack_clr[top] = cur | (left << 21) | (right << 42);
            stack_valid_pos[top] = valid_pos;
            top += 32;
        }

        partial_sum[tid] = sum;
    }
}

long long cuda_n_queens(int N, int level) {
    long long sum = 0;
    vector<int> tot;

    partial_n_queens(N, 0, 0, 0, tot, level);

    if (N & 0x1) {
        partial_n_queens_for_odd(N, 0, 0, 0, tot, level);
    }

    long long cnt = tot.size() / 3;
    vector<long long> partial_sum(cnt);

    hipDeviceSetCacheConfig(hipFuncCachePreferShared);

    int *cuda_tot;
    long long *cuda_partial_sum;
    CU_SAFE_CALL(hipMalloc(&cuda_tot, sizeof(int) * cnt * 3));
    CU_SAFE_CALL(hipMalloc(&cuda_partial_sum, sizeof(long long) * cnt));

    CU_SAFE_CALL(hipMemcpy(cuda_tot, tot.data(), sizeof(int) * cnt * 3, hipMemcpyHostToDevice));
    CU_SAFE_CALL(hipMemset(cuda_partial_sum, 0, sizeof(long long) * cnt));

    dim3 dimBlock(CU1DBLOCK);
    dim3 dimGrid(get_block_size(cnt, CU1DBLOCK));

    printf("total size %lld, block size %d, grid size %d\n", cnt, CU1DBLOCK, get_block_size(cnt, CU1DBLOCK));

    n_queens<<<dimGrid, dimBlock>>>(N, cuda_tot, cuda_partial_sum, cnt);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("kernel error: %s\n", hipGetErrorString(err));
    }

    CU_SAFE_CALL(hipMemcpy(partial_sum.data(), cuda_partial_sum, sizeof(long long) * cnt, hipMemcpyDeviceToHost));

    for (long long i = 0; i < cnt; i++) {
        sum += partial_sum[i] * 2;
    }

    CU_SAFE_CALL(hipFree(cuda_tot));
    CU_SAFE_CALL(hipFree(cuda_partial_sum));

    return sum;
}
